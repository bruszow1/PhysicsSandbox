#include "hip/hip_runtime.h"
#include "draw.h"
#include "BitMap.h"
#include "movement.h"
#include "utils.h"

//constexpr double PI = 3.1415926535897932384626433832795028841971693993751058209;
//constexpr double FOV = 150 * PI / 180.0;
//constexpr double MIN_DOUBLE_THRESHOLD = .00001;


void draw_shape(Shape *shape, unsigned int triangle_index, Pixel *bitmap, double **depth_map)
{
    double current_x = shape->origin[0];
    double current_y = shape->origin[1];
    double current_z = shape->origin[2];

    unsigned int triangle_origin_vertex = shape->triangles[triangle_index].origin_vector_index;

    double shape_z = shape->origin[2];
    double z_scaling = IMAGE_WIDTH / (IMAGE_WIDTH + 2 * shape_z * tan(PI / 2.0 - FOV / 2.0));
    current_x = shape->origin[0] + shape->vector_array[triangle_origin_vertex][0] * z_scaling;
    current_y = shape->origin[1] + shape->vector_array[triangle_origin_vertex][1] * z_scaling;
    current_z = shape->origin[2] + shape->vector_array[triangle_origin_vertex][2] * z_scaling;
    double perception_x_side_change = shape_z / tan(FOV / 2.0);
    double perception_x_range = IMAGE_WIDTH + 2 * perception_x_side_change;
    double corrected_x_origin = (shape->origin[0] + perception_x_side_change) / perception_x_range * IMAGE_WIDTH;
    current_x = corrected_x_origin + shape->vector_array[triangle_origin_vertex][0] * z_scaling;

    int y_image_size = IMAGE_HEIGHT;
    double perception_y_side_change = shape_z / tan(FOV / 2.0) * 9.0 / 16.0;
    double perception_y_range = y_image_size + 2 * perception_y_side_change;
    double corrected_y_origin = (shape->origin[1] + perception_y_side_change) / perception_y_range * IMAGE_HEIGHT;
    current_y = corrected_y_origin + shape->vector_array[triangle_origin_vertex][1] * z_scaling;

    // loop over each side vector
    for (unsigned int side_index = 0; side_index < 3; side_index++) {
        unsigned int vector_index = shape->triangles[triangle_index].side_vector_index + side_index;
        double vector_x = shape->vector_array[vector_index][0];
        double vector_y = shape->vector_array[vector_index][1];
        double vector_z = shape->vector_array[vector_index][2];

        // normalize vector increments to max size .5; prevents skipping pixels in bitmap
        double increment_sum = abs(vector_x) + abs(vector_y) + abs(vector_z);
        double x_increment = (increment_sum == 0) ? 0 : vector_x / increment_sum / 2.0;
        double y_increment = (increment_sum == 0) ? 0 : vector_y / increment_sum / 2.0;
        double z_increment = (increment_sum == 0) ? 0 : vector_z / increment_sum / 2.0;

        double dest_x = current_x + vector_x * z_scaling;
        double dest_y = current_y + vector_y * z_scaling;
        double dest_z = current_z + vector_z * z_scaling;
        bool x_reached = false, y_reached = false, z_reached = false;
        bool no_color = shape->triangles[triangle_index].side_colors[side_index][0] == 0 &&
                        shape->triangles[triangle_index].side_colors[side_index][1] == 0 &&
                        shape->triangles[triangle_index].side_colors[side_index][2] == 0;
        while (!x_reached || !y_reached || !z_reached) {
            const int x_pixel = (int) current_x;
            const int y_pixel = (int) current_y;

            // do not color if index out of bounds or color is black (same as background; can cover drawn lines)
            if (!no_color && current_x >= 0 && current_x <= IMAGE_WIDTH && current_y >= 0 &&
                current_y <= IMAGE_HEIGHT && current_z > -1) {

                const unsigned int pixel_index = y_pixel * IMAGE_WIDTH + x_pixel;
                if (depth_map[y_pixel][x_pixel] < 0 || current_z < depth_map[y_pixel][x_pixel]) {
                    unsigned int bitmap_index = y_pixel * IMAGE_WIDTH + x_pixel;
                    bitmap[bitmap_index].blue = shape->triangles[triangle_index].side_colors[side_index][0];
                    bitmap[bitmap_index].green = shape->triangles[triangle_index].side_colors[side_index][1];
                    bitmap[bitmap_index].red = shape->triangles[triangle_index].side_colors[side_index][2];

                    depth_map[y_pixel][x_pixel] = current_z;
                }
            }

            // update progress booleans
            x_reached = abs(x_increment) < 0.01 ||
                        (x_increment > 0 ? trunc(current_x) >= trunc(dest_x) : trunc(current_x) <= trunc(dest_x));
            y_reached = abs(y_increment) < 0.01 ||
                        (y_increment > 0 ? trunc(current_y) >= trunc(dest_y) : trunc(current_y) <= trunc(dest_y));
            z_reached = abs(z_increment) < .01 ||
                        (z_increment > 0 ? trunc(current_z) >= trunc(dest_z) : trunc(current_z) <= trunc(dest_z));

            // increment coords until new pixel is reached or z-axis destination is reached
            do {
                current_x += x_increment;
                current_y += y_increment;
                current_z += z_increment;
                z_reached = abs(z_increment) < .01 ||
                            (z_increment > 0 ? trunc(current_z) >= trunc(dest_z) : trunc(current_z) <= trunc(dest_z));
            } while (trunc(current_x) == x_pixel && trunc(current_y) == y_pixel && !z_reached);
        }
    }
}

void draw_texture(Shape *shape, unsigned int tri_index, Pixel *bitmap, double **depth_map)
{
    // find triangle's origin
    double shape_z = shape->origin[2];
    double z_scaling = IMAGE_WIDTH / (IMAGE_WIDTH + 2 * shape_z * tan(PI / 2.0 - FOV / 2.0));

    double calc_shape_origin_coords[3];
    unsigned int origin_vertex_index = shape->triangles[tri_index].origin_vector_index;
    calc_shape_origin_coords[0] = shape->origin[0] + shape->vector_array[origin_vertex_index][0] * z_scaling;
    calc_shape_origin_coords[1] = shape->origin[1] + shape->vector_array[origin_vertex_index][1] * z_scaling;
    calc_shape_origin_coords[2] = shape->origin[2] + shape->vector_array[origin_vertex_index][2] * z_scaling;

    // map shape coord origin to bitmap coord origin
    double perception_x_side_change = shape_z / tan(FOV / 2.0);
    double perception_x_range = IMAGE_WIDTH + 2 * perception_x_side_change;
    double corrected_x_origin = (shape->origin[0] + perception_x_side_change) / perception_x_range * IMAGE_WIDTH;
    calc_shape_origin_coords[0] = corrected_x_origin + shape->vector_array[origin_vertex_index][0] * z_scaling;

    int y_image_size = IMAGE_HEIGHT;
    double perception_y_side_change = shape_z / tan(FOV / 2.0) * 9.0 / 16.0;
    double perception_y_range = y_image_size + 2 * perception_y_side_change;
    double corrected_y_origin = (shape->origin[1] + perception_y_side_change) / perception_y_range * IMAGE_HEIGHT;
    calc_shape_origin_coords[1] = corrected_y_origin + shape->vector_array[origin_vertex_index][1] * z_scaling;

    // save scaled triangle sides in local variable
    double side_1[3];
    double side_2[3];
    double side_3[3];
    unsigned int side_vertex_index = shape->triangles[tri_index].side_vector_index;
    for (unsigned int i = 0; i < 3; i++) {
        side_1[i] = shape->vector_array[side_vertex_index][i] * z_scaling;
        side_2[i] = shape->vector_array[side_vertex_index + 1][i] * z_scaling;
        side_3[i] = shape->vector_array[side_vertex_index + 2][i] * z_scaling;

    }

    // return if triangle's x/y area is 0
    if ((abs(side_1[0]) <= MIN_DOUBLE_THRESHOLD && abs(side_1[1]) <= MIN_DOUBLE_THRESHOLD) ||
        (abs(side_2[0]) <= MIN_DOUBLE_THRESHOLD && abs(side_2[1]) <= MIN_DOUBLE_THRESHOLD) ||
        (abs(side_3[0]) <= MIN_DOUBLE_THRESHOLD && abs(side_3[1]) <= MIN_DOUBLE_THRESHOLD)) {
        return;
    }

    // find bounding box for triangle
    double coord_x_limit[2];
    double coord_y_limit[2];
    double coord_z_limit[2];
    triangle_min_max_coords(side_1, side_2, side_3, coord_x_limit, coord_y_limit, coord_z_limit,
                            calc_shape_origin_coords, 1);

    // z-coord must be ignored for triangle bounds check but needed to determine closer shape if overlap exists
    double true_x_z = side_1[2];
    double true_y_z = side_2[2];
    double true_z_z = side_3[2];
    side_1[2] = 0;
    side_2[2] = 0;
    side_3[2] = 0;

    // find normal vector of triangle's plane
    double normal_vector[3];
    cross(normal_vector, side_1[0], side_1[1], true_x_z, side_2[0], side_2[1], true_y_z);

    // loop over bounding box
    for (int bitmap_y_index = coord_y_limit[0]; bitmap_y_index < coord_y_limit[1]; bitmap_y_index++) {
        for (int bitmap_x_index = coord_x_limit[0]; bitmap_x_index < coord_x_limit[1]; bitmap_x_index++) {
            if (plane_point_in_triangle(side_1, side_2, side_3, bitmap_x_index, bitmap_y_index,
                calc_shape_origin_coords[2], calc_shape_origin_coords)) {

                // find z coordinate
                double z_coord_bitmap = (-1 * normal_vector[0] * (bitmap_x_index - calc_shape_origin_coords[0])
                                        - normal_vector[1] * (bitmap_y_index - calc_shape_origin_coords[1]))
                                        / (normal_vector[2]);
                z_coord_bitmap += calc_shape_origin_coords[2];

                int texture_bitmap_index = 0; // placeholder pending mapping adjustments for bitmaps

                bool valid_coord = bitmap_x_index >= 0 && bitmap_y_index >= 0 &&
                                    bitmap_x_index < IMAGE_WIDTH && bitmap_y_index < IMAGE_HEIGHT;
                if (valid_coord && (depth_map[bitmap_y_index][bitmap_x_index] < 0 ||
                                    z_coord_bitmap < depth_map[bitmap_y_index][bitmap_x_index])) {

                    // update bitmap pixel
                    unsigned int bitmap_index = bitmap_y_index * IMAGE_WIDTH + bitmap_x_index;
                    bitmap[bitmap_index].blue = shape->triangles[tri_index].bitmap[texture_bitmap_index].blue;
                    bitmap[bitmap_index].green = shape->triangles[tri_index].bitmap[texture_bitmap_index].green;
                    bitmap[bitmap_index].red = shape->triangles[tri_index].bitmap[texture_bitmap_index].red;

                    // update depth map
                    depth_map[bitmap_y_index][bitmap_x_index] = z_coord_bitmap;
                }
            }
        }
    }
}
