#include "bitmap.h"
#include "movement.h"
#include "draw.h"


int main(int argc, char *argv[]) {
    int iteration_count = 50;

    Pixel *bitmap = new Pixel[IMAGE_WIDTH * IMAGE_HEIGHT];

    // create shapes
    Shape **shape_arr;
    unsigned int shape_count = 2;
    shape_arr = new Shape*[shape_count];

    shape_arr[0] = new Shape(12, 12, 9);
    double origin1[] = {200, 175, 425};
    double velocity1[] = {5, 0, 0};
    double angular_velocity1[] = {0, 0, 0};
    assemble_cube45(shape_arr[0], origin1, 300, velocity1, angular_velocity1);

    shape_arr[1] = new Shape(12, 12, 9);
    double origin2[] = {650, 300, 500};
    double velocity2[] = {0, 0, 0};
    double angular_velocity2[] = {0, 0, 0};
    assemble_cube(shape_arr[1], origin2, 300, velocity2, angular_velocity2);

    draw_shapes(bitmap, shape_arr, shape_count);
    const std::string file_name_start = "initial_state.bmp";
    write_bitmap_to_file(bitmap, file_name_start);

    for (unsigned int i = 0; i < iteration_count; i++) {
        model_movement(shape_arr, shape_count, 0, 0, 0, 0, 1);
    }

    draw_shapes(bitmap, shape_arr, shape_count);
    const std::string file_name_end = "final_state.bmp";
    write_bitmap_to_file(bitmap, file_name_end);

    delete[] shape_arr;
    return 0;
}