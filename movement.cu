#include "hip/hip_runtime.h"
#include <algorithm>
#include <cmath>
#include "hipblas.h"
#include "nppcore.h"
#include "nppi.h"
#include <thrust/transform.h>
#include <thrust/extrema.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sequence.h>
#include <thrust/execution_policy.h>
#include "movement.h"
#include "shape.h"
#include "utils.h"
#include "draw.h"

hipblasStatus_t rotate_vector(double *target, double *output, double x_rotate, double y_rotate, double z_rotate, hipblasHandle_t *handle) {

    const double rot_matrix[9] = { cos(x_rotate) * cos(y_rotate), sin(y_rotate), -sin(x_rotate) * cos(y_rotate),
        -cos(x_rotate) * sin(y_rotate) * cos(z_rotate) + sin(x_rotate) * sin(z_rotate), cos(y_rotate) * cos(z_rotate),
        sin(x_rotate) * sin(y_rotate) * cos(z_rotate) + cos(x_rotate) * sin(z_rotate),
        cos(x_rotate) * sin(y_rotate) * sin(z_rotate) + sin(x_rotate) * cos(z_rotate), -cos(y_rotate) * sin(z_rotate),
        -sin(x_rotate) * sin(y_rotate) * cos(z_rotate) * sin(z_rotate) + cos(x_rotate) * cos(z_rotate)};

    // alpha = 1 so matrix is unchanged, beta = 0 b/c the + C portion isn't needed
    const double alpha = 1.0;
    const double beta = 0;

    // allocate memory
    double *gpu_rot_matrix;
    hipMalloc((void**) &gpu_rot_matrix, 9 * sizeof(double));
    double *gpu_x;
    hipMalloc(&gpu_x, 3 * sizeof(double));
    double *gpu_y;
    hipMalloc(&gpu_y, 3 * sizeof(double));

    // copy matrix and vectors to GPU memory
    hipMemcpy(gpu_rot_matrix, rot_matrix, 9 * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(gpu_x, target, 3 * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(gpu_y, output, 3 * sizeof(double), hipMemcpyHostToDevice);

    hipblasStatus_t status = hipblasDgemv(*handle, HIPBLAS_OP_N, 3, 3, &alpha, gpu_rot_matrix, 3, gpu_x, 1, &beta, gpu_y, 1);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        printf("Cublas error: %d\n", status);
    }

    // copy result from GPU memory
    hipMemcpy(output, gpu_y, 3 * sizeof(double), hipMemcpyDeviceToHost);

    // clean up memory
    hipFree(gpu_rot_matrix);
    hipFree(gpu_x);
    hipFree(gpu_y);
    return status;
}

void shift_vector(double *origin_vector, double x_move, double y_move, double z_move)
{
    origin_vector[0] += x_move;
    origin_vector[1] += y_move;
    origin_vector[2] += z_move;
}

void rotate_vector(double target[3], double output[3], double x_rotate, double y_rotate, double z_rotate)
{
    double init_x = target[0];
    double init_y = target[1];
    double init_z = target[2];

    // Use 3d rotation matrix
    output[0] = init_x * cos(x_rotate) * cos(y_rotate) + init_y * (sin(x_rotate) * sin(z_rotate) - cos(x_rotate) * sin(y_rotate) * cos(z_rotate))
                + init_z * (sin(x_rotate) * cos(z_rotate) + cos(x_rotate) * sin(y_rotate) * sin(z_rotate));
    output[1] = init_x * sin(y_rotate) + init_y * cos(y_rotate) * cos(z_rotate) - init_z * cos(y_rotate) * sin(z_rotate);
    output[2] = -1 * init_x * sin(x_rotate) * cos(y_rotate) + init_y * (sin(x_rotate) * sin(y_rotate) * cos(z_rotate) + cos(x_rotate) * sin(z_rotate))
                + init_z * (cos(x_rotate) * cos(z_rotate) - sin(x_rotate) * sin(y_rotate) * sin(z_rotate));
}

void rotate_around(Shape *shape, double x_rotate, double y_rotate, double z_rotate, double x_fixed, double y_fixed, double z_fixed, hipblasHandle_t *handle)
{
    // get baseline orientation of input coords and shape origin
    double shape_axis_vector[3] = { x_fixed - shape->origin[0], y_fixed - shape->origin[1], z_fixed - shape->origin[2] };

    const double rot_matrix[9] = { cos(x_rotate) * cos(y_rotate), sin(y_rotate), -sin(x_rotate) * cos(y_rotate),
        -cos(x_rotate) * sin(y_rotate) * cos(z_rotate) + sin(x_rotate) * sin(z_rotate), cos(y_rotate) * cos(z_rotate),
        sin(x_rotate) * sin(y_rotate) * cos(z_rotate) + cos(x_rotate) * sin(z_rotate),
        cos(x_rotate) * sin(y_rotate) * sin(z_rotate) + sin(x_rotate) * cos(z_rotate), -cos(y_rotate) * sin(z_rotate),
        -sin(x_rotate) * sin(y_rotate) * cos(z_rotate) * sin(z_rotate) + cos(x_rotate) * cos(z_rotate)};

    // alpha = 1 so matrix is unchanged, beta = 0 b/c the + C portion isn't needed
    const double alpha = 1.0;
    const double beta = 0;

    // allocate GPU memory
    unsigned int vector_count = 3 * shape->triangle_count + shape->vertex_count + 1;
    unsigned int double_count = 3 * vector_count;
    double *gpu_rot_matrix;
    hipMalloc((void**) &gpu_rot_matrix, 9 * sizeof(double));
    double *gpu_B;
    hipMalloc(&gpu_B, double_count * sizeof(double));
    double *gpu_C;
    hipMalloc(&gpu_C, double_count * sizeof(double));

    // copy vectors and rotation matrix to GPU memory
    hipMemcpy(gpu_rot_matrix, rot_matrix, 9 * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(gpu_B, shape_axis_vector, 3 * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(gpu_B + 3, shape->vector_array[0], (double_count - 3) * sizeof(double), hipMemcpyHostToDevice);

    hipblasStatus_t status = hipblasDgemm(*handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 3, vector_count, 3,  &alpha, gpu_rot_matrix, 3, gpu_B, 3, &beta, gpu_C, 3);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        printf("Cublas error: %d\n", status);
    }

    // copy results from GPU memory
    hipMemcpy(shape_axis_vector, gpu_C, 3 * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(&shape->vector_array[0], gpu_C + 3, (double_count - 3) * sizeof(double), hipMemcpyDeviceToHost);

    // clean up GPU memory
    hipFree(gpu_rot_matrix);
    hipFree(gpu_B);
    hipFree(gpu_C);

    // shift shape so target point is at original position
    double x_correction = x_fixed - (shape_axis_vector[0] + shape->origin[0]);
    double y_correction = y_fixed - (shape_axis_vector[1] + shape->origin[1]);
    double z_correction = z_fixed - (shape_axis_vector[2] + shape->origin[2]);
    shift_vector(shape->origin, x_correction, y_correction, z_correction);
}

void adjust_movement(Shape *shape, double initial_timestep, double current_timestep)
{
    double delta_t = current_timestep - initial_timestep;
    double x_displacement = shape->velocity[0] * delta_t;
    double y_displacement = shape->velocity[1] * delta_t;
    double z_displacement = shape->velocity[2] * delta_t;

    // move shape to correct coordinates
    shift_vector(shape->origin, x_displacement, y_displacement, z_displacement);
}

void adjust_rotation(Shape *shape, double initial_timestep, double current_timestep, hipblasHandle_t *handle)
{
    double delta_t = current_timestep - initial_timestep;

    double x_angular_displacement = shape->angular_velocity[0] * delta_t;
    double y_angular_displacement = shape->angular_velocity[1] * delta_t;
    double z_angular_displacement = shape->angular_velocity[2] * delta_t;

    rotate_around(shape, x_angular_displacement, y_angular_displacement, z_angular_displacement,
                shape->origin[0], shape->origin[1], shape->origin[2], handle);
}

// find thetas required to rotate vec1 onto vec2
void angle_between_vectors(double *vec1, double *vec2, double *result) {
    result[0] = 0, result[1] = 0, result[2] = 0;

    if (vec1[0] != vec2[0]) { // angle in x/z plane
        double vec1_x_theta = 0;
        if (!(vec1[0] == 0 && vec1[2] == 0)) {
            vec1_x_theta = vec1[0] == 0 ? (vec1[2] < 0 ? -PI / 2.0 : PI / 2.0) : atan(vec1[2] / vec1[0]);
        }
        double vec2_x_theta = 0;
        if (!(vec2[0] == 0 && vec2[2] == 0)) {
            vec2_x_theta = vec2[0] == 0 ? (vec2[2] < 0 ? -PI / 2.0 : PI / 2.0) : atan(vec2[2] / vec2[0]);
        }

        // move to quadrants 2/3 if x negative
        vec1_x_theta += vec1[0] < 0 ? PI : 0;
        vec2_x_theta += vec2[0] < 0 ? PI : 0;

        // use only positive angles
        vec1_x_theta += vec1_x_theta < 0 ? 2 * PI : 0;
        vec2_x_theta += vec2_x_theta < 0 ? 2 * PI : 0;

        // issues with 0 angle b/c -0 is treated as negative
        if (!((vec1_x_theta == 0 || vec1_x_theta == PI) && (vec2_x_theta == 0 || vec2_x_theta == PI))) {
            result[0] = vec2_x_theta - vec1_x_theta;
        }
    }
    if (vec1[1] != vec2[1]) { // angle in x/y plane
        double vec1_y_theta = 0;
        if (!(vec1[0] == 0 && vec1[1] == 0)) {
            vec1_y_theta = vec1[0] == 0 ? (vec1[1] < 0 ? -PI / 2.0 : PI / 2.0) : atan(vec1[1] / vec1[0]);
        }
        double vec2_y_theta = 0;
        if (!(vec2[0] == 0 && vec2[1] == 0)) {
            vec2_y_theta = vec2[0] == 0 ? (vec2[1] < 0 ? -PI / 2.0 : PI / 2.0) : atan(vec2[1] / vec2[0]);
        }

        // move to quadrants 2/3 if x negative
        vec1_y_theta += vec1[0] < 0 ? PI : 0;
        vec2_y_theta += vec2[0] < 0 ? PI : 0;

        // use only positive angles
        vec1_y_theta += vec1_y_theta < 0 ? 2 * PI : 0;
        vec2_y_theta += vec2_y_theta < 0 ? 2 * PI : 0;

        // issues with 0 angle b/c -0 is treated as negative
        if (!((vec1_y_theta == 0 || vec1_y_theta == PI) && (vec2_y_theta == 0 || vec2_y_theta == PI))) {
            result[1] = vec2_y_theta - vec1_y_theta;
        }
    }
    if (vec1[2] != vec2[2]) { // angle in y/z plane
        double vec1_z_theta = 0;
        if (!(vec1[1] == 0 && vec1[2] == 0)) {
            vec1_z_theta = vec1[2] == 0 ? (vec1[1] < 0 ? -PI / 2.0 : PI / 2.0) : atan(vec1[1] / vec1[2]);
        }
        double vec2_z_theta = 0;
        if (!(vec2[1] == 0 && vec2[2] == 0)) {
            vec2_z_theta = vec2[2] == 0 ? (vec2[1] < 0 ? -PI / 2.0 : PI / 2.0) : atan(vec2[1] / vec2[2]);
        }

        // move to quadrants 2/3 if x negative
        vec1_z_theta += vec1[2] < 0 ? PI : 0;
        vec2_z_theta += vec2[2] < 0 ? PI : 0;

        // use only positive angles
        vec1_z_theta += vec1_z_theta < 0 ? 2 * PI : 0;
        vec2_z_theta += vec2_z_theta < 0 ? 2 * PI : 0;

        // issues with 0 angle b/c -0 is treated as negative
        if (!((vec1_z_theta == 0 || vec1_z_theta == PI) && (vec2_z_theta == 0 || vec2_z_theta == PI))) {
            result[2] = vec2_z_theta - vec1_z_theta;
        }
    }
}

void convert_angular_to_linear(Shape *shape, double *shape1_vel_angular, double *linear_vel, double *collision_coords) {
    // find direction; tangent of y-axis to collision point
    double shape1_to_collision[3];
    double s_1_collision_sum = 0;
    for (unsigned int i = 0; i < 3; i++) {
        shape1_to_collision[i] = collision_coords[i] - shape->origin[i];
        s_1_collision_sum += abs(shape1_to_collision[i]);
    }
    double x_sum = abs(shape1_to_collision[0]) + abs(shape1_to_collision[2]) == 0 ? 1 : abs(shape1_to_collision[0]) + abs(shape1_to_collision[2]);
    double s1_angular_x[3] = { 0, 0, 0 };
    s1_angular_x[0] = abs(shape1_to_collision[2]) / x_sum * (shape1_vel_angular[0] >= 0 ? 1 : -1);
    s1_angular_x[2] = abs(shape1_to_collision[0]) / x_sum * (shape1_vel_angular[0] >= 0 ? -1 : 1);
    // angular to linear
    double x_radius = sqrt(pow(s1_angular_x[0] * x_sum, 2) + pow(s1_angular_x[2] * x_sum, 2)); //radius at height
    s1_angular_x[0] *= x_radius * abs(shape1_vel_angular[0]);
    s1_angular_x[2] *= x_radius * abs(shape1_vel_angular[0]);
    linear_vel[0] += s1_angular_x[0];
    linear_vel[2] += s1_angular_x[2];

    // y angular to linear
    double y_sum = abs(shape1_to_collision[0]) + abs(shape1_to_collision[1]) == 0 ? 1 : abs(shape1_to_collision[0]) + abs(shape1_to_collision[1]);
    double s1_angular_y[3] = { 0, 0, 0 };
    s1_angular_y[0] = abs(shape1_to_collision[1]) / y_sum * (shape1_vel_angular[1] >= 0 ? -1 : 1);
    s1_angular_y[1] = abs(shape1_to_collision[0]) / y_sum * (shape1_vel_angular[1] >= 0 ? 1 : -1);
    // angular to linear
    double y_radius = sqrt(pow(s1_angular_y[0] * y_sum, 2) + pow(s1_angular_y[1] * y_sum, 2));
    s1_angular_y[0] *= y_radius * abs(shape1_vel_angular[1]);
    s1_angular_y[1] *= y_radius * abs(shape1_vel_angular[1]);
    linear_vel[0] += s1_angular_y[0];
    linear_vel[1] += s1_angular_y[1];

    // z angular to linear
    double z_sum = abs(shape1_to_collision[1]) + abs(shape1_to_collision[2]) == 0 ? 1 : abs(shape1_to_collision[1]) + abs(shape1_to_collision[2]);
    double s1_angular_z[3] = { 0, 0, 0 };
    s1_angular_z[1] = abs(shape1_to_collision[2]) / z_sum * (shape1_vel_angular[2] >= 0 ? -1 : 1);
    s1_angular_z[2] = abs(shape1_to_collision[1]) / z_sum * (shape1_vel_angular[2] >= 0 ? 1 : -1);
    double z_radius = sqrt(pow(s1_angular_z[1] * z_sum, 2) + pow(s1_angular_z[2] * z_sum, 2));
    s1_angular_z[1] *= z_radius * abs(shape1_vel_angular[2]);
    s1_angular_z[2] *= z_radius * abs(shape1_vel_angular[2]);
    linear_vel[1] += s1_angular_z[1];
    linear_vel[2] += s1_angular_z[2];
}

void convert_linear_to_angular(Shape *shape, double *shape1_linear, double *angular_vel, double *collision_coords) {
    double linear_unit[3] = { shape1_linear[0], shape1_linear[1], shape1_linear[2] };
    double linear_sum = 0;
    for (unsigned int i = 0; i < 3; i++) {
        linear_sum += abs(shape1_linear[i]);
    }
    for (unsigned int i = 0; i < 3; i++) {
        linear_unit[i] /= linear_sum == 0 ? 1 : linear_sum;
    }
    double linear_mag = sqrt(pow(shape1_linear[0], 2) + pow(shape1_linear[1], 2) + pow(shape1_linear[2], 2));

    double shape1_to_collision[3];
    double s_1_collision_sum = 0;
    for (unsigned int i = 0; i < 3; i++) {
        shape1_to_collision[i] = collision_coords[i] - shape->origin[i];
        s_1_collision_sum += abs(shape1_to_collision[i]);
    }

    double x_direction;
    if ((collision_coords[2] >= 0 && shape1_linear[0] >= 0) || (collision_coords[2] < 0 && shape1_linear[0] < 0)) {
        x_direction = 1;
    } else {
        x_direction = -1;
    }
    double x_sum = abs(shape1_to_collision[0]) + abs(shape1_to_collision[2]) == 0 ? 1 : abs(shape1_to_collision[0]) + abs(shape1_to_collision[2]);
    double s1_angular_x[3] = { 0, 0, 0 };
    s1_angular_x[0] = abs(shape1_to_collision[2]) / x_sum * (shape1_linear[0] >= 0 ? 1 : -1);
    s1_angular_x[2] = abs(shape1_to_collision[0]) / x_sum * (shape1_linear[0] >= 0 ? -1 : 1);
    double overlap = dot(linear_unit, s1_angular_x);
    // angular to linear
    double x_radius = sqrt(pow(shape1_to_collision[0], 2) + pow(shape1_to_collision[2], 2)); //radius at height
    angular_vel[0] += x_radius == 0 ? 0 : linear_mag * overlap / (2 * PI * x_radius) * x_direction;

    // y angular to linear
    double y_direction;
    if ((collision_coords[1] >= 0 && shape1_linear[0] < 0) || (collision_coords[1] < 0 && shape1_linear[0] >= 0)) {
        y_direction = 1;
    } else {
        y_direction = -1;
    }
    double y_sum = abs(shape1_to_collision[0]) + abs(shape1_to_collision[1]) == 0 ? 1 : abs(shape1_to_collision[0]) + abs(shape1_to_collision[1]);
    double s1_angular_y[3] = { 0, 0, 0 };
    s1_angular_y[0] = abs(shape1_to_collision[1]) / y_sum * (shape1_linear[1] >= 0 ? -1 : 1);
    s1_angular_y[1] = abs(shape1_to_collision[0]) / y_sum * (shape1_linear[1] >= 0 ? 1 : -1);
    overlap = dot(linear_unit, s1_angular_y);
    // angular to linear
    double y_radius = sqrt(pow(shape1_to_collision[0], 2) + pow(shape1_to_collision[1], 2)); //radius at height
    angular_vel[1] += y_radius == 0 ? 0 : linear_mag * overlap / (2 * PI * y_radius) * y_direction;

    // z angular to linear
    double z_direction;
    if ((collision_coords[2] >= 0 && shape1_linear[1] <= 0) || (collision_coords[2] < 0 && shape1_linear[1] > 0)) {
        z_direction = 1;
    } else {
        z_direction = -1;
    }
    double z_sum = abs(shape1_to_collision[1]) + abs(shape1_to_collision[2]) == 0 ? 1 : abs(shape1_to_collision[1]) + abs(shape1_to_collision[2]);
    double s1_angular_z[3] = { 0, 0, 0 };
    s1_angular_z[1] = abs(shape1_to_collision[2]) / z_sum * (shape1_linear[2] >= 0 ? -1 : 1);
    s1_angular_z[2] = abs(shape1_to_collision[1]) / z_sum * (shape1_linear[2] >= 0 ? 1 : -1);
    overlap = dot(linear_unit, s1_angular_z);
    // angular to linear
    double z_radius = sqrt(pow(shape1_to_collision[1], 2) + pow(shape1_to_collision[2], 2)); //radius at height
    angular_vel[2] += z_radius == 0 ? 0 : linear_mag * overlap / (2 * PI * z_radius) * z_direction;
}

void calculate_hit(Shape *shape1, Shape *shape2, double *collision_coords,
                    double *shape1_vel_input, double *shape2_vel_input, double *shape1_vel_output,
                    double *shape2_vel_output, double *shape1_vel_output_angular, double *shape2_vel_output_angular) {

    // calculate magnitudes
    double origins_vec[] = { shape2->origin[0] - shape1->origin[0], shape2->origin[1] - shape1->origin[1],
                                shape2->origin[2] - shape1->origin[2] };
    double origin_mag = sqrt(pow(origins_vec[0], 2) + pow(origins_vec[1], 2) + pow(origins_vec[2], 2));
    double shape1_collision_mag = sqrt(pow(collision_coords[0] - shape1->origin[0], 2)
                                             + pow(collision_coords[1] - shape1->origin[1], 2)
                                             + pow(collision_coords[2] - shape1->origin[2], 2));
    double shape2_collision_mag = sqrt(pow(collision_coords[0] - shape2->origin[0], 2)
                                        + pow(collision_coords[1] - shape2->origin[1], 2)
                                        + pow(collision_coords[2] - shape2->origin[2], 2));

    // calculate unit vectors
    double ref_x_vec[] = { 1, 0, 0 } ;
    double origins_contact_sum = abs(origins_vec[0]) + abs(origins_vec[1]) + abs(origins_vec[2]);
    double shape1_vel_sum = abs(shape1_vel_input[0]) + abs(shape1_vel_input[1]) + abs(shape1_vel_input[2]);
    double shape2_vel_sum = abs(shape2_vel_input[0]) + abs(shape2_vel_input[1]) + abs(shape2_vel_input[2]);
    double shape1_unit[3];
    double shape2_unit[3];
    double shape1_to_collision[3];
    double shape2_to_collision[3];
    double shape1_ratio = shape1_collision_mag / origin_mag;
    double shape2_ratio = -1 * shape2_collision_mag / origin_mag;
    double s1_collision_sum = 0;
    double s2_collision_sum = 0;
    for (unsigned int i = 0; i < 3; i++) {
        shape1_unit[i] = shape1_vel_sum == 0 ? 0 : shape1_vel_input[i] / shape1_vel_sum;
        shape2_unit[i] = shape2_vel_sum == 0 ? 0 : shape2_vel_input[i] / shape2_vel_sum;

        shape1_to_collision[i] = origins_vec[i] * shape1_ratio;
        s1_collision_sum += abs(shape1_to_collision[i]);
        shape2_to_collision[i] = origins_vec[i] * shape2_ratio;
        s2_collision_sum += abs(shape2_to_collision[i]);

        origins_vec[i] /= origins_contact_sum == 0 ? 1 : origins_contact_sum;
    }
    for (unsigned int i = 0; i < 3; i++) {
        shape1_to_collision[i] /= s1_collision_sum == 0 ? 1 : s1_collision_sum;
        shape2_to_collision[i] /= s2_collision_sum == 0 ? 1 : s2_collision_sum;
    }

    // find angles between unit vectors
    double s1_point_thetas[3];
    angle_between_vectors(shape1_to_collision, ref_x_vec, s1_point_thetas);
    double s2_point_thetas[3];
    angle_between_vectors(shape2_to_collision, ref_x_vec, s2_point_thetas);
    double s1_thetas[3];
    angle_between_vectors(shape1_unit, origins_vec, s1_thetas);
    double s2_thetas[3];
    angle_between_vectors(shape2_unit, origins_vec, s2_thetas);

    // correct tangent vector is the one that points toward the velocity vector
    double s1_thetas_tangent[3] = { 0, 0, 0 };
    for (unsigned int i = 0; i < 3; i++) {
        if (s1_thetas[i] > 0) {
            s1_thetas_tangent[i] = -PI / 2.0;
        } else if (s1_thetas[i] < 0) {
            s1_thetas_tangent[i] = PI / 2.0;
        }
    }
    if (s1_thetas[0] == 0 && s1_thetas[1] == 0 && s1_thetas[2] == 0) {
        s1_thetas_tangent[0] = PI / 2.0;
        s1_thetas_tangent[1] = PI / 2.0;
        s1_thetas_tangent[2] = PI / 2.0;
    }
    double tangent_vec[3];
    rotate_vector(origins_vec, tangent_vec, s1_thetas_tangent[0], s1_thetas_tangent[1], s1_thetas_tangent[2]);

    // calculate dot products for scaling
    double shape_1_aligned_dot = dot(shape1_unit, origins_vec);
    double shape_1_tangent_dot = dot(shape1_unit, tangent_vec);
    double shape_2_aligned_dot = dot(shape2_unit, origins_vec);
    double shape_2_tangent_dot = dot(shape2_unit, tangent_vec);

    // rotate velocity vectors to origin and tangent components
    double shape1_aligned_vector[3] = {shape1_vel_input[0], shape1_vel_input[1], shape1_vel_input[2]};
    rotate_vector(shape1_aligned_vector, shape1_aligned_vector, s1_thetas[0], s1_thetas[1], s1_thetas[2]);
    double shape2_aligned_vector[3] = {shape2_vel_input[0], shape2_vel_input[1], shape2_vel_input[2]};
    rotate_vector(shape2_aligned_vector, shape2_aligned_vector, s1_thetas[0], s1_thetas[1], s1_thetas[2]);
    double shape1_tangent_vector[3] = {shape1_vel_input[0], shape1_vel_input[1], shape1_vel_input[2]};
    rotate_vector(shape1_tangent_vector, shape1_tangent_vector, s1_thetas[0] + s1_thetas_tangent[0],
                s1_thetas[1] + s1_thetas_tangent[1], s1_thetas[2] + s1_thetas_tangent[2]);
    double shape2_tangent_vector[3] = {shape2_vel_input[0], shape2_vel_input[1], shape2_vel_input[2]};
    rotate_vector(shape2_tangent_vector, shape2_tangent_vector, -(s1_thetas[0] + s1_thetas_tangent[0]),
        -(s1_thetas[1] + s1_thetas_tangent[1]), -(s1_thetas[2] + s1_thetas_tangent[2]));

    // scale vectors
    for (unsigned int i = 0; i < 3; i++) {
        shape1_aligned_vector[i] *= abs(shape_1_aligned_dot);
        shape2_aligned_vector[i] *= abs(shape_2_aligned_dot);
        shape1_tangent_vector[i] *= abs(shape_1_tangent_dot);
        shape2_tangent_vector[i] *= abs(shape_2_tangent_dot);
    }

    // angular velocity calculated in terms of linear velocity by using positive x-axis as reference point
    double shape1_normalized_tangent[3] = {shape1_vel_input[0], shape1_vel_input[1], shape1_vel_input[2]};
    double shape2_normalized_tangent[3] = {shape2_vel_input[0], shape2_vel_input[1], shape2_vel_input[2]};
    rotate_vector(shape1_tangent_vector, shape1_normalized_tangent, s1_point_thetas[0], s1_point_thetas[1], s1_point_thetas[2]);
    rotate_vector(shape2_tangent_vector, shape2_normalized_tangent, -s2_point_thetas[0], -s2_point_thetas[1], -s2_point_thetas[2]);

    // calculate velocities using conservation of momentum
    double shape1_lin_angular[3];
    double shape2_lin_angular[3];
    for (unsigned int i = 0; i < 3; i++) {
        shape1_vel_output[i] += (shape1->mass - shape2->mass) / (shape1->mass + shape2->mass) * shape1_aligned_vector[i]
                                    + 2 * shape2->mass / (shape1->mass + shape2->mass) * shape2_aligned_vector[i];
        shape2_vel_output[i] += 2 * shape1->mass / (shape1->mass + shape2->mass) * shape1_aligned_vector[i]
                                    + (shape2->mass - shape1->mass) / (shape1->mass + shape2->mass) * shape2_aligned_vector[i];

        shape1_vel_output[i] += shape1_tangent_vector[i];
        shape2_vel_output[i] += shape2_tangent_vector[i];

        shape1_lin_angular[i] = (shape1->mass - shape2->mass) / (shape1->mass + shape2->mass) * shape1_normalized_tangent[i]
        + 2 * shape2->mass / (shape1->mass + shape2->mass) * shape2_normalized_tangent[i];
        shape2_lin_angular[i] = 2 * shape1->mass / (shape1->mass + shape2->mass) * shape1_normalized_tangent[i]
        + (shape2->mass - shape1->mass) / (shape1->mass + shape2->mass) * shape2_normalized_tangent[i];
    }

    // using positive x-axis as reference, x velocity corresponds to z rotation
    shape1_vel_output_angular[0] += shape1_lin_angular[2] / shape1_collision_mag;
    shape2_vel_output_angular[0] += shape2_lin_angular[2] / shape2_collision_mag * -1;

    shape1_vel_output_angular[1] += shape1_lin_angular[1] / shape1_collision_mag;
    shape2_vel_output_angular[1] += shape2_lin_angular[1] / shape2_collision_mag * -1;

    // using positive x-axis as reference, z velocity corresponds to x rotation
    shape1_vel_output_angular[2] += shape1_lin_angular[0] / shape1_collision_mag;
    shape2_vel_output_angular[2] += shape2_lin_angular[0] / shape2_collision_mag * -1;
}

void update_velocities(Shape *shape1, Shape *shape2, double *collision_coords) {
    // calculate linear velocity contribution
    double shape1_vel[3] = {shape1->velocity[0], shape1->velocity[1], shape1->velocity[2]};
    double shape2_vel[3] = {shape2->velocity[0], shape2->velocity[1], shape2->velocity[2]};
    double shape1_vel_angular[3] = { shape1->angular_velocity[0], shape1->angular_velocity[1], shape1->angular_velocity[2] };
    double shape2_vel_angular[3] = { shape2->angular_velocity[0], shape2->angular_velocity[1], shape2->angular_velocity[2] };

    double shape_1_vel_output[3] = {0, 0, 0};
    double shape_2_vel_output[3] = {0, 0, 0};
    double shape_1_vel_output_angular[3] = {0, 0, 0};
    double shape_2_vel_output_angular[3] = {0, 0, 0};

    calculate_hit(shape1, shape2, collision_coords, shape1_vel, shape2_vel, shape_1_vel_output,
        shape_2_vel_output, shape_1_vel_output_angular, shape_2_vel_output_angular);

    double s1_vel[] = { 0, 0, 0 };
    double s2_vel[] = { 0, 0, 0 };
    convert_angular_to_linear(shape1, shape1_vel_angular, s1_vel, collision_coords);
    convert_angular_to_linear(shape2, shape2_vel_angular, s2_vel, collision_coords);

    double dummy_output[3] = {0.0, 0.0, 0.0}; // placeholder; need to prevent shape's angular velocity from turning into linear
    double dummy_output2[3] = {0.0, 0.0, 0.0};
    calculate_hit(shape1, shape2, collision_coords, s1_vel, s2_vel, dummy_output,
        dummy_output2, shape_1_vel_output_angular, shape_2_vel_output_angular);

    double s1_vel_output[] = { 0, 0, 0};
    double s2_vel_output[] = { 0, 0, 0};
    convert_linear_to_angular(shape1, dummy_output, s1_vel_output, collision_coords);
    convert_linear_to_angular(shape2, dummy_output2, s2_vel_output, collision_coords);
    for (unsigned int i = 0; i < 3; i++) {
        shape_1_vel_output_angular[i] += s1_vel_output[i];
        shape_2_vel_output_angular[i] += s2_vel_output[i];
    }

    // copy results to shapes
    for (unsigned int i = 0; i < 3; i++) {
        shape1->velocity[i] = shape_1_vel_output[i];
        shape2->velocity[i] = shape_2_vel_output[i];
        shape1->angular_velocity[i] = shape_1_vel_output_angular[i];
        shape2->angular_velocity[i] = shape_2_vel_output_angular[i];
    }
}

int triangle_intersection(Shape *shape, unsigned int tri_index, Shape *shape2, unsigned int tri_index2, double *tri1_origin, double *tri2_origin, double *coords)
{
    int intersection_found = 0;
    unsigned int intersection_count = 0;
    unsigned int shape1_side1 = shape->triangles[tri_index].side_vector_index;

    // calculate shape_2 triangle plane
    unsigned int shape2_side1 = shape2->triangles[tri_index2].side_vector_index;
    double normal_vector[3];
    cross(normal_vector,
        shape2->vector_array[shape2_side1][0], shape2->vector_array[shape2_side1][1], shape2->vector_array[shape2_side1][2],
        shape2->vector_array[shape2_side1 + 1][0], shape2->vector_array[shape2_side1 + 1][1], shape2->vector_array[shape2_side1 + 1][2]);

    double current_origin[3] = { tri1_origin[0], tri1_origin[1], tri1_origin[2] };
    for (unsigned int tri_outer = 0; tri_outer < 3; tri_outer++) {
        // find point where triangle side intersects shape2 face
        double t_val = -1 * current_origin[0] * normal_vector[0] + normal_vector[0] * tri2_origin[0] - current_origin[1] * normal_vector[1]
                        + normal_vector[1] * tri2_origin[1] - current_origin[2] * normal_vector[2] + normal_vector[2] * tri2_origin[2];
        double t_val_divisor = normal_vector[0] * shape->vector_array[shape1_side1 + tri_outer][0] +
                                normal_vector[1] * shape->vector_array[shape1_side1 + tri_outer][1] +
                                normal_vector[2] * shape->vector_array[shape1_side1 + tri_outer][2];
        if (t_val_divisor == 0) {
            continue;
        }
        t_val /= t_val_divisor;
        double x_point = t_val * shape->vector_array[shape1_side1 + tri_outer][0] + current_origin[0];
        double y_point = t_val * shape->vector_array[shape1_side1 + tri_outer][1] + current_origin[1];
        double z_point = t_val * shape->vector_array[shape1_side1 + tri_outer][2] + current_origin[2];

        // verify intersection point is in both triangles
        bool point_in_s1 = plane_point_in_triangle(shape->vector_array[shape1_side1], shape->vector_array[shape1_side1 + 1],
                                                    shape->vector_array[shape1_side1 + 2], x_point, y_point, z_point, tri1_origin);
        bool point_in_s2 = plane_point_in_triangle(shape2->vector_array[shape2_side1], shape2->vector_array[shape2_side1 + 1],
                                                    shape2->vector_array[shape2_side1 + 2], x_point, y_point, z_point, tri2_origin);
        if (point_in_s2 && point_in_s1) {
            if (intersection_found <= 1) {
                intersection_found = 1;
            }

            coords[0] = x_point;
            coords[1] = y_point;
            coords[2] = z_point;

            // check if shape1 side is parallel to shape2 face
            double x_mult = normal_vector[0] == 0 ? 0 : abs(shape->vector_array[shape1_side1 + tri_outer][0] / normal_vector[0]);
            double y_mult = normal_vector[1] == 0 ? 0 : abs(shape->vector_array[shape1_side1 + tri_outer][1] / normal_vector[1]);
            double z_mult = normal_vector[2] == 0 ? 0 : abs(shape->vector_array[shape1_side1 + tri_outer][2] / normal_vector[2]);
            bool xy_pass = x_mult == 0 || y_mult == 0 || x_mult == y_mult;
            bool xz_pass = x_mult == 0 || z_mult == 0 || x_mult == z_mult;
            bool yz_pass = y_mult == 0 || z_mult == 0 || y_mult == z_mult;
            if (xy_pass && yz_pass && xz_pass) {
                if (intersection_found <= 2) {
                    intersection_found = 2;
                }
                intersection_count++;

                // parallel vector means collision is line; use midpoint as collision point
                coords[3] = current_origin[0] + shape->vector_array[shape1_side1 + tri_outer][0] / 2.0;
                coords[4] = current_origin[1] + shape->vector_array[shape1_side1 + tri_outer][1] / 2.0;
                coords[5] = current_origin[2] + shape->vector_array[shape1_side1 + tri_outer][2] / 2.0;
            }
        }
        current_origin[0] += shape->vector_array[shape1_side1 + tri_outer][0];
        current_origin[1] += shape->vector_array[shape1_side1 + tri_outer][1];
        current_origin[2] += shape->vector_array[shape1_side1 + tri_outer][2];
    }

    if (intersection_count == 3) { // 3 intersections means plane is parallel
        intersection_found = 3;
        // use average of vertices as collision point
        coords[6] = 0;
        coords[7] = 0;
        coords[8] = 0;
        for (unsigned int i = 0; i < 3; i++) {
            for (unsigned int j = 0; j < 3; j++) {
                coords[6 + j] += (current_origin[j] + shape->vector_array[shape1_side1 + i][j]) / 3.0;
                current_origin[j] += shape->vector_array[shape1_side1 + i][j];
            }
        }
    }
    return intersection_found;
}

int calculate_plane_lines(Shape *shape, unsigned int tri_index, Shape *shape2, unsigned int tri_index2, double *coords)
{
    unsigned int shape1_origin_index = shape->triangles[tri_index].origin_vector_index;
    unsigned int shape2_origin_index = shape2->triangles[tri_index2].origin_vector_index;

    double shape1_origin[3];
    double shape2_origin[3];
    for (unsigned int i = 0; i < 3; i++) {
        shape1_origin[i] = shape->origin[i] + shape->vector_array[shape1_origin_index][i];
        shape2_origin[i] = shape2->origin[i] + shape2->vector_array[shape2_origin_index][i];
    }

    int shape1_pass = triangle_intersection(shape, tri_index, shape2, tri_index2, shape1_origin, shape2_origin, coords);
    int shape2_pass = triangle_intersection(shape2, tri_index2, shape, tri_index, shape2_origin, shape1_origin, coords);

    if (shape1_pass > 0 || shape2_pass > 0) {
        return max(shape1_pass, shape2_pass);
    }
    return 0;
}

// functor for thrust
struct tri_collision_check {
    Shape *shape1;
    Shape *shape2;
    double *collision_coords;

    tri_collision_check(Shape *shape1, Shape *shape2, double *collision_coords) : shape1(shape1), shape2(shape2), collision_coords(collision_coords) {}

    int operator()(const int& input1, const int& input2) {
        unsigned int tri_index1 = input1 / 12;
        unsigned int tri_index2 = input2 % 12;
        int collision = calculate_plane_lines(shape1, tri_index1, shape2, tri_index2, collision_coords);
        return collision;
    }
};

bool check_collision(Shape *shape1, Shape *shape2)
{
    // find shape bounding box
    double shape_1_x_min_max[2] = {shape1->origin[0], shape1->origin[0]};
    double shape_1_y_min_max[2] = {shape1->origin[1], shape1->origin[1]};
    double shape_1_z_min_max[2] = {shape1->origin[2], shape1->origin[2]};
    shape_min_max_coords(shape1, shape_1_x_min_max, shape_1_y_min_max, shape_1_z_min_max);

    double shape_2_x_min_max[2] = {shape2->origin[0], shape2->origin[0]};
    double shape_2_y_min_max[2] = {shape2->origin[1], shape2->origin[1]};
    double shape_2_z_min_max[2] = {shape2->origin[2], shape2->origin[2]};
    shape_min_max_coords(shape2, shape_2_x_min_max, shape_2_y_min_max, shape_2_z_min_max);

    bool impossible_x_intersection = (shape_1_x_min_max[1] < shape_2_x_min_max[0] || shape_1_x_min_max[0] > shape_2_x_min_max[1]);
    bool impossible_y_intersection = (shape_1_y_min_max[1] < shape_2_y_min_max[0] || shape_1_y_min_max[0] > shape_2_y_min_max[1]);
    bool impossible_z_intersection = (shape_1_z_min_max[1] < shape_2_z_min_max[0] || shape_1_z_min_max[0] > shape_2_z_min_max[1]);
    if (impossible_x_intersection || impossible_y_intersection || impossible_z_intersection) {
        return false;
    }

    thrust::host_vector<int> shape_1_vec(shape1->triangle_count * shape2->triangle_count);
    thrust::host_vector<int> shape_2_vec(shape1->triangle_count * shape2->triangle_count);
    thrust::host_vector<int> output_vec(shape1->triangle_count * shape2->triangle_count);

    thrust::sequence(thrust::host, shape_1_vec.begin(), shape_1_vec.end());
    thrust::sequence(thrust::host, shape_2_vec.begin(), shape_2_vec.end());

    double collision_coords[9];
    thrust::transform(thrust::host, shape_1_vec.begin(), shape_1_vec.end(), shape_2_vec.begin(),
                        output_vec.begin(), tri_collision_check(shape1, shape2, collision_coords));

    int collision = *thrust::max_element(output_vec.begin(), output_vec.end());
    if (collision > 0) {
        double passed_collision[3];
        for (unsigned int i = 0; i < 3; i++) {
            passed_collision[i] = collision_coords[(collision - 1) * 3 + i];
        }

        update_velocities(shape1, shape2, passed_collision);
        printf("collision detected\n");

        // shift_vector(shape1->origin, -200, 0, 0);

        return true;
    }
    return false;
}

void model_movement(Shape **input_shapes, unsigned int shape_count, int x_rotate, int y_rotate, int z_rotate,
                    double initial_timestep, double final_timestep) {

    // rotate around origin
    double axis_x = input_shapes[0]->origin[0];
    double axis_y = input_shapes[0]->origin[1];
    double axis_z = input_shapes[0]->origin[2];

    hipblasHandle_t cnpHandle;
    hipblasStatus_t status = hipblasCreate(&cnpHandle);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        printf("Cublas error: %d\n", status);
    }
    rotate_around(input_shapes[0], degrees_to_radians(x_rotate), degrees_to_radians(y_rotate), degrees_to_radians(z_rotate),
                    axis_x, axis_y, axis_z, &cnpHandle);

    for (unsigned int i = 0; i < shape_count; i++) {
        adjust_movement(input_shapes[i], initial_timestep, final_timestep);
        adjust_rotation(input_shapes[i], initial_timestep, final_timestep, &cnpHandle);
    }

    check_collision(input_shapes[0], input_shapes[1]);

    hipblasDestroy(cnpHandle);
}

void draw_shapes(Pixel *bitmap, Shape **input_shapes, unsigned int shape_count) {
    double side_len = 10; // placeholder

    int texture_size = (int) (side_len * side_len);
    Pixel *blue_bitmap = new Pixel[texture_size];
    for (int i = 0; i < texture_size; i++) {
        blue_bitmap[i].blue = 255;
        blue_bitmap[i].green = 0;
        blue_bitmap[i].red = 0;
    }
    Pixel *green_bitmap = new Pixel[texture_size];
    for (int i = 0; i < texture_size; i++) {
        green_bitmap[i].blue = 0;
        green_bitmap[i].green = 255;
        green_bitmap[i].red = 0;
    }
    Pixel *red_bitmap = new Pixel[texture_size];
    for (int i = 0; i < texture_size; i++) {
        red_bitmap[i].blue = 0;
        red_bitmap[i].green = 0;
        red_bitmap[i].red = 255;
    }
    Pixel *pink_bitmap = new Pixel[texture_size];
    for (int i = 0; i < texture_size; i++) {
        pink_bitmap[i].blue = 255;
        pink_bitmap[i].green = 0;
        pink_bitmap[i].red = 255;
    }
    Pixel *yellow_bitmap = new Pixel[texture_size];
    for (int i = 0; i < texture_size; i++) {
        yellow_bitmap[i].blue = 0;
        yellow_bitmap[i].green = 255;
        yellow_bitmap[i].red = 255;
    }
    Pixel *cyan_bitmap = new Pixel[texture_size];
    for (int i = 0; i < texture_size; i++) {
        cyan_bitmap[i].blue = 255;
        cyan_bitmap[i].green = 255;
        cyan_bitmap[i].red = 0;
    }

    double **depth_map = new double*[IMAGE_HEIGHT];
    for (int i = 0; i < IMAGE_HEIGHT; i++) {
        depth_map[i] = new double[IMAGE_WIDTH];
    }
    for (int i = 0; i < (IMAGE_HEIGHT); i++) {
        for (int j = 0; j < (IMAGE_WIDTH); j++) {
            depth_map[i][j] = -1.0;
        }
    }

    // zero passed bitmap
    // allocate GPU memory
    Npp8u *srcPtr;
    Npp8u *dstPtr;
    hipMalloc(&srcPtr, IMAGE_WIDTH * IMAGE_HEIGHT * sizeof(uint8_t) * 3);
    hipMalloc(&dstPtr, IMAGE_WIDTH * IMAGE_HEIGHT * sizeof(uint8_t) * 3);

    // copy bitmap to GPU memory
    hipMemcpy(dstPtr, bitmap, IMAGE_WIDTH * IMAGE_HEIGHT * sizeof(uint8_t) * 3, hipMemcpyHostToDevice);

    // Use NPP to subtract the base bitmap from itself
    int bytes_per_row = sizeof(uint8_t) * 3 * IMAGE_WIDTH;
    NppiSize roiSize;
    roiSize.width = IMAGE_WIDTH;
    roiSize.height = IMAGE_HEIGHT;
    hipDeviceSynchronize();

    NppStatus status = nppiSub_8u_C3IRSfs(dstPtr, bytes_per_row, dstPtr, bytes_per_row, roiSize, 0);
    if (status != NPP_SUCCESS) {
        printf("Cublas error: %d\n", status);
    }

    // copy result from GPU memory
    hipMemcpy(bitmap, dstPtr, IMAGE_WIDTH * IMAGE_HEIGHT * sizeof(uint8_t) * 3, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    // free GPU memory
    hipFree(dstPtr);
    hipFree(srcPtr);

    // iterate over input shape array
    for (unsigned int i = 0; i < shape_count; i++) {
        Shape *cube = input_shapes[i];

        // placeholder pending mapping changes for bitmap
        cube->triangles[0].bitmap = green_bitmap;
        cube->triangles[1].bitmap = green_bitmap;
        cube->triangles[2].bitmap = blue_bitmap;
        cube->triangles[3].bitmap = blue_bitmap;
        cube->triangles[4].bitmap = red_bitmap;
        cube->triangles[5].bitmap = red_bitmap;
        cube->triangles[6].bitmap = pink_bitmap;
        cube->triangles[7].bitmap = pink_bitmap;
        cube->triangles[8].bitmap = yellow_bitmap;
        cube->triangles[9].bitmap = yellow_bitmap;
        cube->triangles[10].bitmap = cyan_bitmap;
        cube->triangles[11].bitmap = cyan_bitmap;

        for (unsigned int triangle_index = 0; triangle_index < cube->triangle_count; triangle_index++) {
            draw_shape(cube, triangle_index, bitmap, depth_map);
        }
        for (unsigned int texture_index = 0; texture_index < cube->triangle_count; texture_index++) {
            draw_texture(cube, texture_index, bitmap, depth_map);
        }
    }

    // clean up memory
    delete[] blue_bitmap;
    delete[] green_bitmap;
    delete[] red_bitmap;
    delete[] pink_bitmap;
    delete[] yellow_bitmap;
    delete[] cyan_bitmap;
    for (int i = 0; i < (IMAGE_HEIGHT); i++) {
        delete[] depth_map[i];
    }
    delete[] depth_map;
}
