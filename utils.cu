#include "hip/hip_runtime.h"
#include <algorithm>
#include <cmath>
#include "utils.h"
#include "movement.h"

double degrees_to_radians(double degrees)
{
    return degrees * (PI / 180.0);
}

// finds the coords of a shape's bounding box
void shape_min_max_coords(Shape* shape, double* x_min_max, double* y_min_max, double* z_min_max)
{
    for (unsigned int i = 0; i < shape->vertex_count; i++) {
        double *current_vector = shape->vector_array[i];

        x_min_max[0] = min(x_min_max[0], current_vector[0] + shape->origin[0]);
        x_min_max[1] = max(x_min_max[1], current_vector[0] + shape->origin[0]);

        y_min_max[0] = min(y_min_max[0], current_vector[1] + shape->origin[1]);
        y_min_max[1] = max(y_min_max[1], current_vector[1] + shape->origin[1]);

        z_min_max[0] = min(z_min_max[0], current_vector[2] + shape->origin[2]);
        z_min_max[1] = max(z_min_max[1], current_vector[2] + shape->origin[2]);
    }
}

// finds the bounding box of a triangle
void triangle_min_max_coords(double *side1, double *side2, double *side3, double *x_min_max, double *y_min_max, double *z_min_max, double *origin_coord, double scale_factor)
{
    // set initial min/max values to starting coord
    for (unsigned int i = 0; i < 2; i++) {
        x_min_max[i] = origin_coord[0];
        y_min_max[i] = origin_coord[1];
        z_min_max[i] = origin_coord[2];
    }

    double current_coord[3] = { origin_coord[0], origin_coord[1], origin_coord[2] };
    double *tri_side_ptrs[3] = { side1, side2, side3 };

    // side3 not checked; endpoint is assumed to be equal to origin
    for (unsigned int side_index = 0; side_index < 2; side_index++) {
        // find next coord by moving to vector's endpoint
        for (unsigned int i = 0; i < 3; i++) {
            current_coord[i] += tri_side_ptrs[side_index][i] * scale_factor;
        }

        // compare coord to existing value
        x_min_max[0] = min(x_min_max[0], current_coord[0]);
        x_min_max[1] = max(x_min_max[1], current_coord[0]);
        y_min_max[0] = min(y_min_max[0], current_coord[1]);
        y_min_max[1] = max(y_min_max[1], current_coord[1]);
        z_min_max[0] = min(z_min_max[0], current_coord[2]);
        z_min_max[1] = max(z_min_max[1], current_coord[2]);
    }
}

// find cross product of two vectors
void cross(double* input, double px1, double py1, double pz1, double px2, double py2, double pz2) {
    input[0] = py1 * pz2 - pz1 * py2;
    input[1] = pz1 * px2 - px1 * pz2;
    input[2] = px1 * py2 - py1 * px2;
}

// find dot product of two vectors
double dot(double *parr1, double *parr2) {
    return parr1[0] * parr2[0] + parr1[1] * parr2[1] + parr1[2] * parr2[2];
}

// helper function that checks if a point is on the same side of two vectors using the right-hand rule
bool same_side(double px1, double py1, double pz1, double px2, double py2, double pz2, double ax, double ay, double az, double bx, double by, double bz) {
    double cross1[3];
    double cross2[3];
    cross(cross1, bx - ax, by - ay, bz - az, px1 - ax, py1 - ay, pz1 - az);
    cross(cross2, bx - ax, by - ay, bz - az, px2 - ax, py2 - ay, pz2 - az);

    if (dot(cross1, cross2) >= 0) {
        return true;
    }
    return false;
}

bool plane_point_in_triangle(double *side1, double *side2, double *side3, double x, double y, double z, double *tri_origin) {
    // calculate coords of triangle vertices
    double ax = tri_origin[0];
    double ay = tri_origin[1];
    double az = tri_origin[2];

    double bx = ax + side1[0];
    double by = ay + side1[1];
    double bz = az + side1[2];

    double cx = bx + side2[0];
    double cy = by + side2[1];
    double cz = bz + side2[2];

    // check if point is on the same side for each pair of triangle sides
    bool side1_pass = same_side(x, y, z, ax, ay, az, bx, by, bz, cx, cy, cz);
    bool side2_pass = same_side(x, y, z, bx, by, bz, ax, ay, az, cx, cy, cz);
    bool side3_pass = same_side(x, y, z, cx, cy, cz, ax, ay, az, bx, by, bz);

    return side1_pass && side2_pass && side3_pass;

}
